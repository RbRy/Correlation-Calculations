#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "H5Cpp.h"
#include <vector>
#include <H5Exception.h>
#include <map>
#include <omp.h>

// The following lines must be located BEFORE '#include <mex.h>'
#ifdef _MSC_VER
#define DLL_EXPORT_SYM __declspec(dllexport)
#else
#define DLL_EXPORT_SYM
#endif
#include <mex.h>

const int max_tags_length = 500000;
const int max_clock_tags_length = 5000;
const int max_channels = 3;
const size_t return_size = 3;
const int file_block_size = 4;
const double tagger_resolution = 82.3e-12;

struct shotData {
	bool file_load_completed;
	std::vector<short int> channel_list;
	std::map<short int, short int> channel_map;
	std::vector<long long int> start_tags;
	std::vector<long long int> end_tags;
	std::vector<long long int> photon_tags;
	std::vector<long long int> clock_tags;
	std::vector<std::vector<long long int>> sorted_photon_tags;
	std::vector<std::vector<long int>> sorted_photon_bins;
	std::vector<std::vector<long long int>> sorted_clock_tags;
	std::vector<std::vector<long int>> sorted_clock_bins;
	std::vector<long int> sorted_photon_tag_pointers;
	std::vector<long int> sorted_clock_tag_pointers;

	shotData() : sorted_photon_tags(max_channels, std::vector<long long int>(max_tags_length, 0)), sorted_photon_bins(max_channels, std::vector<long int>(max_tags_length, 0)), sorted_photon_tag_pointers(max_channels, 0), sorted_clock_tags(2, std::vector<long long int>(max_clock_tags_length, 0)), sorted_clock_bins(2, std::vector<long int>(max_clock_tags_length, 0)), sorted_clock_tag_pointers(2, 0) {}
};

struct gpuData {
	long int *numer_gpu;
	long int *denom_gpu;
	long int *photon_bins_gpu;
	long int *start_and_end_clocks_gpu;
	int *max_bin_gpu, *pulse_spacing_gpu, *max_pulse_distance_gpu, *photon_bins_length_gpu;
	int *offset_gpu;
};

__global__ void calculateNumeratorGPU_g3(long int *numer, long int *photon_bins, long int *start_and_end_clocks, int *max_bin, int *pulse_spacing, int *max_pulse_distance, int *offset, int *photon_bins_length, int num_channels, int shot_file_num) {
	//Get numerator step to work on
	int id_x = threadIdx.x;
	int block_x = blockIdx.x;
	int block_size_x = blockDim.x;
	int id_y = threadIdx.y;
	int block_y = blockIdx.y;
	int block_size_y = blockDim.y;

	//Check we're not calculating something out of range
	if ((block_x * block_size_x + id_x < *max_bin * 2 + 1) && (block_y * block_size_y + id_y < *max_bin * 2 + 1)) {
		int tau_1 = block_x * block_size_x + id_x - (*max_bin);
		int tau_2 = block_y * block_size_y + id_y - (*max_bin);
		for (int channel_1 = 0; channel_1 < num_channels; channel_1++) {
			for (int channel_2 = channel_1 + 1; channel_2 < num_channels; channel_2++) {
				for (int channel_3 = channel_2 + 1; channel_3 < num_channels; channel_3++) {
					int i = 0;
					int j = 0;
					int k = 0;
					int running_tot = 0;
					while ((i < photon_bins_length[channel_1 + shot_file_num * max_channels]) && (j < photon_bins_length[channel_2 + shot_file_num * max_channels]) && (k < photon_bins_length[channel_3 + shot_file_num * max_channels])) {
						int dummy_i = 0;
						int dummy_j = 0;
						int dummy_k = 0;

						int out_window = (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] < (*max_bin + *max_pulse_distance * *pulse_spacing + start_and_end_clocks[0 + shot_file_num * 2])) || (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] > (start_and_end_clocks[1 + shot_file_num * 2] - (*max_bin + *max_pulse_distance * *pulse_spacing)));
						//Chan_1 > chan_2
						int c1_g_c2 = !out_window && (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] >(photon_bins[offset[channel_2 + shot_file_num * max_channels] + j] - tau_1));
						//Chan_1 > chan_3
						int c1_g_c3 = !out_window && (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] >(photon_bins[offset[channel_3 + shot_file_num * max_channels] + k] - tau_2));
						////Chan_1 < chan_2
						//int c1_l_c2 = !out_window && (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] < (photon_bins[offset[channel_2 + shot_file_num * max_channels] + j] - tau_1));
						////Chan_1 < chan_3
						//int c1_l_c3 = !out_window && (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] > (photon_bins[offset[channel_3 + shot_file_num * max_channels] + k] - tau_2));
						//Chan_1 == chan_2
						int c1_e_c2 = !out_window && (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] == (photon_bins[offset[channel_2 + shot_file_num * max_channels] + j] - tau_1));
						//Chan_1 == chan_3
						int c1_e_c3 = !out_window && (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] == (photon_bins[offset[channel_3 + shot_file_num * max_channels] + k] - tau_2));

						//Increment i if we're outside the window of interest
						dummy_i = out_window;

						//Start by using chan_1 as a reference for chan_2 and chan_3 to get them to catch up
						//Increment j if chan_2 < chan_1
						dummy_j += !out_window && c1_g_c2;
						//Increment k if chan_3 < chan_1
						dummy_k += !out_window && c1_g_c3;

						//Now need to deal with situation where chan_1 !> chan_2 && chan_1 !> chan_3
						//First the easy situation where chan_1 == chan_2 == chan_3
						running_tot += !out_window && c1_e_c2 && c1_e_c3;
						dummy_i += !out_window && c1_e_c2 && c1_e_c3;
						dummy_j += !out_window && c1_e_c2 && c1_e_c3;
						dummy_k += !out_window && c1_e_c2 && c1_e_c3;

						//If we haven't incremented dummy_j or dummy_k then by process of elimination dummy_i needs to incremented
						dummy_i += !out_window && !dummy_j && !dummy_k;

						//running_tot += in_window;
						i += dummy_i;
						j += dummy_j;
						k += dummy_k;
					}
					numer[block_x * block_size_x + id_x + (block_y * block_size_y + id_y) * (*max_bin * 2 + 1) + shot_file_num * (*max_bin * 2 + 1) * (*max_bin * 2 + 1)] += running_tot;
				}
			}
		}
	}
}

__global__ void calculateDenominatorGPU_g3(long int *denom, long int *photon_bins, long int *start_and_end_clocks, int *max_bin, int *pulse_spacing, int *max_pulse_distance, int *offset, int *photon_bins_length, int num_channels, int shot_file_num) {
	//Get denominator step to work on
	int id_x = threadIdx.x;
	int block_x = blockIdx.x;
	int block_size_x = blockDim.x;
	int id_y = threadIdx.y;
	int block_y = blockIdx.y;
	int block_size_y = blockDim.y;

	//Check we're not calculating something out of range
	if ((block_x * block_size_x + id_x < *max_pulse_distance * 2 + 1) && (block_y * block_size_y + id_y < *max_pulse_distance * 2 + 1)) {
		int pulse_shift_1 = block_x * block_size_x + id_x - (*max_pulse_distance);
		int pulse_shift_2 = block_y * block_size_y + id_y - (*max_pulse_distance);
		if ((pulse_shift_1 != 0) && (pulse_shift_2 != 0) && (pulse_shift_1 != pulse_shift_2)) {
			for (int channel_1 = 0; channel_1 < num_channels; channel_1++) {
				for (int channel_2 = channel_1 + 1; channel_2 < num_channels; channel_2++) {
					for (int channel_3 = channel_2 + 1; channel_3 < num_channels; channel_3++) {
						int i = 0;
						int j = 0;
						int k = 0;
						int running_tot = 0;
						while ((i < photon_bins_length[channel_1 + shot_file_num * max_channels]) && (j < photon_bins_length[channel_2 + shot_file_num * max_channels]) && (k < photon_bins_length[channel_3 + shot_file_num * max_channels])) {
							int dummy_i = 0;
							int dummy_j = 0;
							int dummy_k = 0;

							int out_window = (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] < (*max_bin + *max_pulse_distance * *pulse_spacing + start_and_end_clocks[0 + shot_file_num * 2])) || (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] > (start_and_end_clocks[1 + shot_file_num * 2] - (*max_bin + *max_pulse_distance * *pulse_spacing)));
							//Chan_1 > chan_2
							int c1_g_c2 = !out_window && (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] >(photon_bins[offset[channel_2 + shot_file_num * max_channels] + j] - pulse_shift_1));
							//Chan_1 > chan_3
							int c1_g_c3 = !out_window && (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] >(photon_bins[offset[channel_3 + shot_file_num * max_channels] + k] - pulse_shift_2));
							////Chan_1 < chan_2
							//int c1_l_c2 = !out_window && (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] < (photon_bins[offset[channel_2 + shot_file_num * max_channels] + j] - pulse_shift_1));
							////Chan_1 < chan_3
							//int c1_l_c3 = !out_window && (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] > (photon_bins[offset[channel_3 + shot_file_num * max_channels] + k] - pulse_shift_2));
							//Chan_1 == chan_2
							int c1_e_c2 = !out_window && (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] == (photon_bins[offset[channel_2 + shot_file_num * max_channels] + j] - pulse_shift_1));
							//Chan_1 == chan_3
							int c1_e_c3 = !out_window && (photon_bins[offset[channel_1 + shot_file_num * max_channels] + i] == (photon_bins[offset[channel_3 + shot_file_num * max_channels] + k] - pulse_shift_2));

							//Increment i if we're outside the window of interest
							dummy_i = out_window;

							//Start by using chan_1 as a reference for chan_2 and chan_3 to get them to catch up
							//Increment j if chan_2 < chan_1
							dummy_j += !out_window && c1_g_c2;
							//Increment k if chan_3 < chan_1
							dummy_k += !out_window && c1_g_c3;

							//Now need to deal with situation where chan_1 !> chan_2 && chan_1 !> chan_3
							//First the easy situation where chan_1 == chan_2 == chan_3
							running_tot += !out_window && c1_e_c2 && c1_e_c3;
							dummy_i += !out_window && c1_e_c2 && c1_e_c3;
							dummy_j += !out_window && c1_e_c2 && c1_e_c3;
							dummy_k += !out_window && c1_e_c2 && c1_e_c3;

							//If we haven't incremented dummy_j or dummy_k then by process of elimination dummy_i needs to incremented
							dummy_i += !out_window && !dummy_j && !dummy_k;

							//running_tot += in_window;
							i += dummy_i;
							j += dummy_j;
							k += dummy_k;
						}
						denom[block_x * block_size_x + id_x + (block_y * block_size_y + id_y) * (*max_pulse_distance * 2 + 1) + shot_file_num * (*max_pulse_distance * 2 + 1) * (*max_pulse_distance * 2 + 1)] += running_tot;
					}
				}
			}
		}
	}
}

//Function grabs all tags and channel list from file
void fileToShotData(shotData *shot_data, char* filename) {
	//Open up file
	H5::H5File file(filename, H5F_ACC_RDONLY);
	//Open up "Tags" group
	H5::Group tag_group(file.openGroup("Tags"));
	//Find out how many tag sets there are, should be 4 if not something is fucky
	hsize_t numTagsSets = tag_group.getNumObjs();
	if (numTagsSets != 4) {
		mexPrintf("There should be 4 sets of Tags, found %i\n", numTagsSets);
		delete filename;
		exit;
	}
	//Read tags to shotData structure
	//First the clock tags
	H5::DataSet clock_dset(tag_group.openDataSet("ClockTags0"));
	H5::DataSpace clock_dspace = clock_dset.getSpace();
	hsize_t clock_length[1];
	clock_dspace.getSimpleExtentDims(clock_length, NULL);
	shot_data->clock_tags.resize(clock_length[0]);
	clock_dset.read(&(*shot_data).clock_tags[0u], H5::PredType::NATIVE_UINT64, clock_dspace);
	clock_dspace.close();
	clock_dset.close();
	//Then start tags
	H5::DataSet start_dset(tag_group.openDataSet("StartTag"));
	H5::DataSpace start_dspace = start_dset.getSpace();
	hsize_t start_length[1];
	start_dspace.getSimpleExtentDims(start_length, NULL);
	shot_data->start_tags.resize(start_length[0]);
	start_dset.read(&(*shot_data).start_tags[0u], H5::PredType::NATIVE_UINT64, start_dspace);
	start_dspace.close();
	start_dset.close();
	//Then end tags
	H5::DataSet end_dset(tag_group.openDataSet("EndTag"));
	H5::DataSpace end_dspace = end_dset.getSpace();
	hsize_t end_length[1];
	end_dspace.getSimpleExtentDims(end_length, NULL);
	shot_data->end_tags.resize(end_length[0]);
	end_dset.read(&(*shot_data).end_tags[0u], H5::PredType::NATIVE_UINT64, end_dspace);
	end_dspace.close();
	end_dset.close();
	//Finally photon tags
	H5::DataSet photon_dset(tag_group.openDataSet("TagWindow0"));
	H5::DataSpace photon_dspace = photon_dset.getSpace();
	hsize_t photon_length[1];
	photon_dspace.getSimpleExtentDims(photon_length, NULL);
	shot_data->photon_tags.resize(photon_length[0]);
	photon_dset.read(&(*shot_data).photon_tags[0u], H5::PredType::NATIVE_UINT64, photon_dspace);
	photon_dspace.close();
	photon_dset.close();
	//And close tags group
	tag_group.close();
	//Open up "Inform" group
	H5::Group inform_group(file.openGroup("Inform"));
	//Grab channel list
	H5::DataSet chan_dset(inform_group.openDataSet("ChannelList"));
	H5::DataSpace chan_dspace = chan_dset.getSpace();
	hsize_t chan_length[1];
	chan_dspace.getSimpleExtentDims(chan_length, NULL);
	shot_data->channel_list.resize(chan_length[0]);
	chan_dset.read(&(*shot_data).channel_list[0u], H5::PredType::NATIVE_UINT16, chan_dspace);
	chan_dspace.close();
	chan_dset.close();
	//Close Inform group
	inform_group.close();
	//Close file
	file.close();

	//Populate channel map
	for (short int i = 0; i < shot_data->channel_list.size(); i++) {
		shot_data->channel_map[shot_data->channel_list[i]] = i;
	}
}

//Reads relevant information for a block of files into shot_block
void populateBlock(std::vector<shotData> *shot_block, std::vector<char *> *filelist, int block_num) {
	//Loop over the block size
	for (int i = 0; i < file_block_size; i++) {
		//Default to assuming the block is corrupted
		(*shot_block)[i].file_load_completed = false;
		//Figure out the file id within the filelist
		int file_id = block_num * file_block_size + i;
		//Check the file_id isn't out of range of the filelist
		if (file_id < filelist->size()) {
			//Try to load file to shot_block
			try {
				fileToShotData(&(*shot_block)[i], (*filelist)[file_id]);
				(*shot_block)[i].file_load_completed = true;
			}
			//Will catch if the file is corrupted, print corrupted filenames to command window
			catch (...) {
				printf("%s appears corrupted\n", (*filelist)[file_id]);
			}
		}
	}
}


//Process the time tags, assigning them to the correct channel, binning them appropriately and removing tags which do not fall in the clock mask
void sortTags(shotData *shot_data) {
	long int i;
	int high_count = 0;
	//Loop over all tags in clock_tags
	for (i = 0; i < shot_data->clock_tags.size(); i++) {
		//Check if clock tag is a high word
		if (shot_data->clock_tags[i] & 1) {
			//Up the high count
			high_count++;
		}
		else {
			//Determine whether it is the rising (start) or falling (end) slope
			int slope = ((shot_data->clock_tags[i] >> 28) & 1);
			//Put tag in appropriate clock tag vector and increment the pointer for said vector
			shot_data->sorted_clock_tags[slope][shot_data->sorted_clock_tag_pointers[slope]] = ((shot_data->clock_tags[i] >> 1) & 0x7FFFFFF) + (high_count << 27) - ((shot_data->start_tags[1] >> 1) & 0x7FFFFFF);
			shot_data->sorted_clock_tag_pointers[slope]++;
		}
	}
	high_count = 0;
	//Clock pointer
	int clock_pointer = 0;
	//Loop over all tags in photon_tags
	for (i = 0; i < shot_data->photon_tags.size(); i++) {
		//Check if photon tag is a high word
		if (shot_data->photon_tags[i] & 1) {
			//Up the high count
			high_count++;
		}
		else {
			//Figure out if it fits within the mask
			long long int time_tag = ((shot_data->photon_tags[i] >> 1) & 0x7FFFFFF) + (high_count << 27) - ((shot_data->start_tags[1] >> 1) & 0x7FFFFFF);
			bool valid = true;
			while (valid) {
				//printf("%i\t%i\t%i\t", time_tag, shot_data->sorted_clock_tags[1][clock_pointer], shot_data->sorted_clock_tags[0][clock_pointer - 1]);
				//Increment dummy pointer if channel tag is greater than current start tag
				if ((time_tag >= shot_data->sorted_clock_tags[1][clock_pointer]) & (clock_pointer < shot_data->sorted_clock_tag_pointers[1])) {
					//printf("up clock pointer\n");
					clock_pointer++;
				}
				//Make sure clock_pointer is greater than 0, preventing an underflow error
				else if (clock_pointer > 0) {
					//Check if tag is lower than previous end tag i.e. startTags[j-1] < channeltags[i] < endTags[j-1]
					if (time_tag <= shot_data->sorted_clock_tags[0][clock_pointer - 1]) {
						//printf("add tag tot data\n");
						//Determine the index for given tag
						int channel_index = shot_data->channel_map.find(((shot_data->photon_tags[i] >> 29) & 7) + 1)->second;
						//Bin tag and assign to appropriate vector
						shot_data->sorted_photon_tags[channel_index][shot_data->sorted_photon_tag_pointers[channel_index]] = time_tag;
						//printf("%i\t%i\t%i\n", channel_index, time_tag, shot_data->sorted_photon_tag_pointers[channel_index]);
						shot_data->sorted_photon_tag_pointers[channel_index]++;
					}
					//Break the valid loop
					valid = false;
				}
				// If tag is smaller than the first start tag
				else {
					valid = false;
				}
			}
		}
	}
}

void tagsToBins(shotData *shot_data, double bin_width) {
	double norm_bin_width = bin_width / tagger_resolution;
	#pragma omp parallel for
	for (int channel = 0; channel < shot_data->sorted_photon_bins.size(); channel++) {
	#pragma omp parallel for
		for (int i = 0; i < shot_data->sorted_photon_tag_pointers[channel]; i++) {
			shot_data->sorted_photon_bins[channel][i] = (long int)ceil(double(shot_data->sorted_photon_tags[channel][i] / norm_bin_width));
		}
	}
	for (int slope = 0; slope <= 1; slope++) {
		#pragma omp parallel for
		for (int i = 0; i < shot_data->sorted_clock_tag_pointers[slope]; i++) {
			shot_data->sorted_clock_bins[slope][i] = (long int)ceil(double(shot_data->sorted_clock_tags[slope][i] / norm_bin_width));
		}
	}
}

//Sorts photons and bins them for each file in a block
void sortAndBinBlock(std::vector<shotData> *shot_block, double bin_width) {
#pragma omp parallel for
	for (int shot_file_num = 0; shot_file_num < file_block_size; shot_file_num++) {
		if ((*shot_block)[shot_file_num].file_load_completed) {
			sortTags(&(*shot_block)[shot_file_num]);
			tagsToBins(&(*shot_block)[shot_file_num], bin_width);
		}
	}
}

void printShotChannelBins(shotData *shot_data, int channel) {
	for (int i = 0; i < shot_data->sorted_photon_tag_pointers[channel]; i++) {
		printf("%i\t%i\t%i\n", i, shot_data->sorted_photon_tags[channel][i], shot_data->sorted_photon_bins[channel][i]);
	}
}

void mexFunction(int nlhs, mxArray* plhs[], int nrgs, const mxArray* prhs[]) {
	//Get list of files to process
	mxArray *cell_element_ptr;
	mwSize total_num_files, buflen;
	//Figure out how many files there are and allocate a vector to hold strings
	total_num_files = mxGetNumberOfElements(prhs[0]);
	std::vector<char *> filelist(total_num_files);
	//Grab filename and stick it into filelist vector
	for (int i = 0; i < total_num_files; i++) {
		cell_element_ptr = mxGetCell(prhs[0], i);
		buflen = mxGetN(cell_element_ptr) * sizeof(mxChar) + 1;
		filelist[i] = (char *)mxMalloc(buflen);
		mxGetString(cell_element_ptr, filelist[i], buflen);
	}

	double *max_time;
	max_time = (double *)mxGetData(prhs[2]);
	double *bin_width;
	bin_width = (double *)mxGetData(prhs[1]);
	double *pulse_spacing;
	pulse_spacing = (double *)mxGetData(prhs[3]);
	int *max_pulse_distance;
	max_pulse_distance = (int *)mxGetData(prhs[4]);

	printf("Bin width\t%f�s\t%fns\t%f�s\t%i\n", *max_time * 1e6, *bin_width * 1e9, *pulse_spacing * 1e6, *max_pulse_distance);

	int max_bin = (int)round(*max_time / *bin_width);
	int bin_pulse_spacing = (int)round(*pulse_spacing / *bin_width);

	//Create our array to hold the denominator and numerator
	plhs[0] = mxCreateNumericMatrix(1, (max_bin * 2 + 1) * (max_bin * 2 + 1), mxINT32_CLASS, mxREAL);
	long int* numer = (long int*)mxGetData(plhs[0]);
	plhs[1] = mxCreateNumericMatrix(1, 1, mxINT32_CLASS, mxREAL);
	long int* denom = (long int*)mxGetData(plhs[1]);
	//Initialise denom and numer to zero
	#pragma omp parallel for
	for (int i = 0; i < (max_bin * 2 + 1) * (max_bin * 2 + 1); i++) {
		numer[i] = 0;
	}
	denom[0] = 0;

	//Figure out how many blocks we need
	int blocks_req;
	if (total_num_files < file_block_size) {
		blocks_req = 1;
	}
	else if ((total_num_files%file_block_size) == 0) {
		blocks_req = total_num_files / file_block_size;
	}
	else {
		blocks_req = total_num_files / file_block_size + 1;
	}
	printf("Processing %i files in %i blocks\n", total_num_files, blocks_req);

	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	//Load some stuff to the GPU we will use permenantly
	//Allocate memory on GPU for various things
	gpuData gpu_data;

	cudaStatus = hipMalloc((void**)&(gpu_data.photon_bins_gpu), max_channels * max_tags_length * file_block_size * sizeof(long int));
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMalloc photon_bins_gpu failed\n");
		mexPrintf("%s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&(gpu_data.offset_gpu), max_channels * file_block_size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMalloc offset_gpu failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&(gpu_data.photon_bins_length_gpu), max_channels * file_block_size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMalloc photon_bins_length_gpu failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&(gpu_data.numer_gpu), (2 * (max_bin)+1) * (2 * (max_bin)+1) * file_block_size * sizeof(long int));
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMalloc numer_gpu failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&(gpu_data.start_and_end_clocks_gpu), 2 * file_block_size * sizeof(long int));
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMalloc start_and_end_clocks_gpu failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&(gpu_data.max_bin_gpu), sizeof(int));
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMalloc max_bin_gpu failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&(gpu_data.pulse_spacing_gpu), sizeof(int));
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMalloc pulse_spacing_gpu failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&(gpu_data.max_pulse_distance_gpu), sizeof(int));
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMalloc max_pulse_distance_gpu failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&(gpu_data.denom_gpu), (*max_pulse_distance * 2 + 1) * (*max_pulse_distance * 2 + 1) * file_block_size * sizeof(long int));
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMalloc max_pulse_distance_gpu failed!\n");
		goto Error;
	}

	//And set some values that are constant across all data
	cudaStatus = hipMemcpy((gpu_data.max_bin_gpu), &max_bin, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMemcpy failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy((gpu_data.pulse_spacing_gpu), &bin_pulse_spacing, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMemcpy failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy((gpu_data.max_pulse_distance_gpu), max_pulse_distance, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMemcpy failed!\n");
		goto Error;
	}

	//Pointer to first photon bin for each channel
	int host_offest_array[max_channels * file_block_size];
	for (int i = 0; i < max_channels * file_block_size; i++) {
		host_offest_array[i] = i * max_tags_length;
	}
	cudaStatus = hipMemcpy((gpu_data.offset_gpu), host_offest_array, max_channels * file_block_size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemset((gpu_data).numer_gpu, 0, (2 * (max_bin)+1) * file_block_size * sizeof(long int));
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMemset failed!\n");
		goto Error;
	}
	cudaStatus = hipMemset((gpu_data).denom_gpu, 0, (*max_pulse_distance * 2 + 1) * (*max_pulse_distance * 2 + 1) * file_block_size * sizeof(long int));
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMemset failed!\n");
		goto Error;
	}

	//Create some streams for us to use for GPU parallelism
	hipStream_t streams[file_block_size];
	for (int i = 0; i < file_block_size; i++) {
		hipStreamCreate(&streams[i]);
	}

	//Figure out how many CUDA blocks to chunk the processing up into for the numerator
	int threads_per_block_dim_numer = 32;
	int cuda_blocks_req_numer = 0;
	if (threads_per_block_dim_numer >= max_bin * 2 + 1) {
		cuda_blocks_req_numer = 1;
	}
	else if (((max_bin * 2 + 1) % threads_per_block_dim_numer) == 0) {
		cuda_blocks_req_numer = (max_bin * 2 + 1) / threads_per_block_dim_numer;
	}
	else {
		cuda_blocks_req_numer = (max_bin * 2 + 1) / threads_per_block_dim_numer + 1;
	}
	dim3 cuda_threads_numer(threads_per_block_dim_numer, threads_per_block_dim_numer);
	dim3 cuda_blocks_numer(cuda_blocks_req_numer, cuda_blocks_req_numer);

	//Figure out how many CUDA blocks to chunk the processing up into for the denominator
	int threads_per_block_dim_denom = (*max_pulse_distance * 2 + 1);
	dim3 cuda_threads_denom(threads_per_block_dim_denom, threads_per_block_dim_denom);
	dim3 cuda_blocks_denom(1, 1);

	//Processes files in blocks
	for (int block_num = 0; block_num < blocks_req; block_num++) {
		//Allocate a vector to hold a block of shot_data
		std::vector<shotData> shot_block(file_block_size);

		//Populate the shot_block with data from file
		populateBlock(&shot_block, &filelist, block_num);

		//Sort tags and convert them to bins
		sortAndBinBlock(&shot_block, *bin_width);
		//printShotChannelBins(&(shot_block[0]), 1);

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		/*cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			mexPrintf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}*/

		//Asyncronously load data to GPU
		for (int shot_file_num = 0; shot_file_num < file_block_size; shot_file_num++) {
			if ((shot_block)[shot_file_num].file_load_completed) {
				int num_channels = (shot_block)[shot_file_num].channel_list.size();
				if (num_channels >= 3) {

					std::vector<long int*> photon_bins;
					long int start_and_end_clocks[2];
					std::vector<int> photon_bins_length;
					photon_bins.resize(max_channels);
					photon_bins_length.resize(max_channels);

					start_and_end_clocks[0] = (shot_block)[shot_file_num].sorted_clock_bins[1][0];
					start_and_end_clocks[1] = (shot_block)[shot_file_num].sorted_clock_bins[0][0];
					for (int i = 0; i < num_channels; i++) {
						photon_bins[i] = &((shot_block)[shot_file_num].sorted_photon_bins[i][0]);
						photon_bins_length[i] = (shot_block)[shot_file_num].sorted_photon_tag_pointers[i];
					}
					//Write photon bins to memory
					int photon_offset = shot_file_num * max_channels * max_tags_length;
					for (int i = 0; i < photon_bins_length.size(); i++) {
						cudaStatus = hipMemcpyAsync((gpu_data).photon_bins_gpu + photon_offset, (photon_bins)[i], (photon_bins_length)[i] * sizeof(long int), hipMemcpyHostToDevice, streams[shot_file_num]);
						if (cudaStatus != hipSuccess) {
							mexPrintf("%i\t%i\n", block_num, shot_file_num);
							mexPrintf("hipMemcpy photon_offset failed! Error message: %s\n", hipGetErrorString(cudaStatus));
							goto Error;
						}
						photon_offset += max_tags_length;
					}

					int clock_offset = shot_file_num * 2;
					//And other parameters
					cudaStatus = hipMemcpyAsync((gpu_data).start_and_end_clocks_gpu + clock_offset, start_and_end_clocks, 2 * sizeof(long int), hipMemcpyHostToDevice, streams[shot_file_num]);
					if (cudaStatus != hipSuccess) {
						mexPrintf("hipMemcpy clock_offset failed!\n");
						goto Error;
					}

					int length_offset = shot_file_num * max_channels;
					//Can't copy vector to cuda easily
					for (int i = 0; i < photon_bins_length.size(); i++) {
						cudaStatus = hipMemcpyAsync((gpu_data).photon_bins_length_gpu + i + length_offset, &((photon_bins_length)[i]), sizeof(int), hipMemcpyHostToDevice, streams[shot_file_num]);
						if (cudaStatus != hipSuccess) {
							mexPrintf("hipMemcpy length_offset failed!\n");
							goto Error;
						}
					}
					
					//Launch numerator calculating kernel for each set of channels
					calculateNumeratorGPU_g3 << <cuda_blocks_numer, cuda_threads_numer, 0, streams[shot_file_num] >> >((gpu_data).numer_gpu, (gpu_data).photon_bins_gpu, (gpu_data).start_and_end_clocks_gpu, (gpu_data).max_bin_gpu, (gpu_data).pulse_spacing_gpu, (gpu_data).max_pulse_distance_gpu, (gpu_data).offset_gpu, (gpu_data).photon_bins_length_gpu, num_channels, shot_file_num);
					//Launch denominator calculating kernel for each set of channels
					calculateDenominatorGPU_g3 << <cuda_blocks_denom, cuda_threads_denom, 0, streams[shot_file_num] >> >((gpu_data).denom_gpu, (gpu_data).photon_bins_gpu, (gpu_data).start_and_end_clocks_gpu, (gpu_data).max_bin_gpu, (gpu_data).pulse_spacing_gpu, (gpu_data).max_pulse_distance_gpu, (gpu_data).offset_gpu, (gpu_data).photon_bins_length_gpu, num_channels, shot_file_num);
					// Check for any errors launching the kernel
					cudaStatus = hipGetLastError();
					if (cudaStatus != hipSuccess) {
						mexPrintf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
						goto Error;
					}
				}
			}
		}
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	//This is to pull the streamed numerator off the GPU
	//Streamed numerator refers to the way the numerator is stored on the GPU where each GPU stream has a seperate numerator
	long int *streamed_numer;
	streamed_numer = (long int *)malloc((2 * (max_bin)+1) * (2 * (max_bin)+1) * file_block_size * sizeof(long int));

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(streamed_numer, (gpu_data).numer_gpu, (2 * (max_bin)+1) * (2 * (max_bin)+1) * file_block_size * sizeof(long int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMemcpy failed!\n");
		free(streamed_numer);
		goto Error;
	}
	//Collapse streamed numerator down to regular numerator
	for (int i = 0; i < file_block_size; i++) {
		for (int j = 0; j < (2 * (max_bin)+1); j++) {
			for (int k = 0; k < (2 * (max_bin)+1); k++) {
				numer[j + k * (2 * (max_bin)+1)] += streamed_numer[j + k * (2 * (max_bin)+1) + i * (2 * (max_bin)+1) * (2 * (max_bin)+1)];
			}
		}
	}

	free(streamed_numer);

	//This is to pull the streamed denominator off the GPU
	//Streamed numerator refers to the way the numerator is stored on the GPU where each GPU stream has a seperate numerator
	long int *streamed_denom;
	streamed_denom = (long int *)malloc((2 * (*max_pulse_distance)+1) * (2 * (*max_pulse_distance)+1) * file_block_size * sizeof(long int));

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(streamed_denom, (gpu_data).denom_gpu, (2 * (*max_pulse_distance)+1) * (2 * (*max_pulse_distance)+1) * file_block_size * sizeof(long int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMemcpy failed!\n");
		free(streamed_denom);
		goto Error;
	}
	//Collapse streamed denominator down to regular denominator
	for (int i = 0; i < (2 * (*max_pulse_distance) + 1) * (2 * (*max_pulse_distance) + 1) * file_block_size; i++) {
		denom[0] += streamed_denom[i];
	}

	free(streamed_denom);

	//Free filenames we malloc'd earlier
	for (int i = 0; i < total_num_files; i++) {
		mxFree(filelist[i]);
	}

	/*cudaStatus = hipFree(gpu_data.max_bin_gpu);
	if (cudaStatus != hipSuccess) {
	mexPrintf("hipDeviceReset failed! %s\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipFree(gpu_data.max_pulse_distance_gpu);
	if (cudaStatus != hipSuccess) {
	mexPrintf("hipDeviceReset failed!\n");
	}
	cudaStatus = hipFree(gpu_data.numer_gpu);
	if (cudaStatus != hipSuccess) {
	mexPrintf("hipDeviceReset failed!\n");
	}
	cudaStatus = hipFree(gpu_data.offset_gpu);
	if (cudaStatus != hipSuccess) {
	mexPrintf("hipDeviceReset failed!\n");
	}
	cudaStatus = hipFree(gpu_data.photon_bins_gpu);
	if (cudaStatus != hipSuccess) {
	mexPrintf("hipDeviceReset failed!\n");
	}
	cudaStatus = hipFree(gpu_data.photon_bins_length_gpu);
	if (cudaStatus != hipSuccess) {
	mexPrintf("hipDeviceReset failed!\n");
	}
	cudaStatus = hipFree(gpu_data.pulse_spacing_gpu);
	if (cudaStatus != hipSuccess) {
	mexPrintf("hipDeviceReset failed!\n");
	}
	cudaStatus = hipFree(gpu_data.start_and_end_clocks_gpu);
	if (cudaStatus != hipSuccess) {
	mexPrintf("hipDeviceReset failed!\n");
	}*/

	//Release CUDA device
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipDeviceReset failed!\n");
	}

Error:
	hipFree((gpu_data.numer_gpu));
	hipFree((gpu_data.offset_gpu));
	hipFree((gpu_data.max_bin_gpu));
	hipFree((gpu_data.pulse_spacing_gpu));
	hipFree((gpu_data.max_pulse_distance_gpu));
	hipFree((gpu_data.photon_bins_length_gpu));
	hipFree(gpu_data.photon_bins_gpu);
	hipFree(gpu_data.start_and_end_clocks_gpu);
	hipDeviceReset();
}